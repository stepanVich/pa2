// includes, cuda
#include <cstdint>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <cudaDefs.h>

hipDeviceProp_t deviceProp = hipDeviceProp_t();

int main(int argc, char* argv[])
{
	initializeCUDA(deviceProp);
}
